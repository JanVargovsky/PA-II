#include "hip/hip_runtime.h"
#include <cudaDefs.h>
#include <time.h>
#include <math.h>

hipError_t error = hipSuccess;
hipDeviceProp_t deviceProp = hipDeviceProp_t();

const unsigned int N = 1 << 20;
const unsigned int MEMSIZE = N * sizeof(unsigned int);
const unsigned int NO_LOOPS = 100;
const unsigned int THREAD_PER_BLOCK = 256;
const unsigned int GRID_SIZE = (N + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK;

void fillData(unsigned int *data, const unsigned int length)
{
	//srand(time(0));
	for (unsigned int i = 0; i < length; i++)
	{
		//data[i]= rand();
		data[i] = 1;
	}
}

void printData(const unsigned int *data, const unsigned int length)
{
	if (data == 0) return;
	for (unsigned int i = 0; i < length; i++)
	{
		printf("%u ", data[i]);
	}
	printf("\n");
}


__global__ void kernel(const unsigned int *a, const unsigned int *b, const unsigned int length, unsigned int *c)
{
	int i = blockDim.x * blockDim.x + threadIdx.x;

	if (i >= length) return;
	//TODO: Vector ADD
	c[i] = a[i] + b[i];
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Tests 1. - single stream, async calling </summary>
///
/// <remarks>	16. 4. 2013. </remarks>
////////////////////////////////////////////////////////////////////////////////////////////////////
void test1()
{
	unsigned int *a, *b, *c;
	unsigned int *da, *db, *dc;

	// paged-locked allocation
	hipHostAlloc((void**)&a, NO_LOOPS * MEMSIZE, hipHostMallocDefault);
	hipHostAlloc((void**)&b, NO_LOOPS * MEMSIZE, hipHostMallocDefault);
	hipHostAlloc((void**)&c, NO_LOOPS * MEMSIZE, hipHostMallocDefault);

	fillData(a, NO_LOOPS * N);
	fillData(b, NO_LOOPS * N);

	// Data chunks on GPU
	hipMalloc((void**)&da, MEMSIZE);
	hipMalloc((void**)&db, MEMSIZE);
	hipMalloc((void**)&dc, MEMSIZE);

	////TODO: create stream
	hipStream_t stream;
	checkCudaErrors(hipStreamCreate(&stream));

	unsigned int dataOffset = 0;
	for (int i = 0; i < NO_LOOPS; i++)
	{
		//TODO:  copy a->da, b->db
		hipMemcpyAsync(da, &a[dataOffset], MEMSIZE, hipMemcpyKind::hipMemcpyHostToDevice, stream);
		hipMemcpyAsync(db, &b[dataOffset], MEMSIZE, hipMemcpyKind::hipMemcpyHostToDevice, stream);

		//TODO:  run the kernel in the stream
		kernel << <GRID_SIZE, THREAD_PER_BLOCK, 0, stream >> > (da, db, MEMSIZE, dc);

		//TODO:  copy dc->c
		hipMemcpyAsync(&c[dataOffset], dc, MEMSIZE, hipMemcpyKind::hipMemcpyDeviceToHost, stream);

		dataOffset += N;
	}

	////TODO: Synchonize stream
	hipStreamSynchronize(stream);
	////TODO: Destroy stream
	checkCudaErrors(hipStreamDestroy(stream));

	printData(a, 10);
	printData(b, 10);
	printData(c, 10);

	hipFree(da);
	hipFree(db);
	hipFree(dc);

	hipHostFree(a);
	hipHostFree(b);
	hipHostFree(c);
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Tests 2. - two streams - depth first approach </summary>
///
/// <remarks>	16. 4. 2013. </remarks>
////////////////////////////////////////////////////////////////////////////////////////////////////
void test2()
{
	//TODO: reuse the source code of above mentioned method test1()
	unsigned int *a, *b, *c;
	unsigned int *da, *db, *dc;

	// paged-locked allocation
	hipHostAlloc((void**)&a, NO_LOOPS * MEMSIZE, hipHostMallocDefault);
	hipHostAlloc((void**)&b, NO_LOOPS * MEMSIZE, hipHostMallocDefault);
	hipHostAlloc((void**)&c, NO_LOOPS * MEMSIZE, hipHostMallocDefault);

	fillData(a, NO_LOOPS * N);
	fillData(b, NO_LOOPS * N);

	// Data chunks on GPU
	hipMalloc((void**)&da, MEMSIZE);
	hipMalloc((void**)&db, MEMSIZE);
	hipMalloc((void**)&dc, MEMSIZE);

	////TODO: create stream
	hipStream_t stream0;
	checkCudaErrors(hipStreamCreate(&stream0));
	hipStream_t stream1;
	checkCudaErrors(hipStreamCreate(&stream1));

	unsigned int dataOffset = 0;
	for (int i = 0; i < NO_LOOPS / 2; i++)
	{
		//TODO:  copy a->da, b->db
		hipMemcpyAsync(da, &a[dataOffset], MEMSIZE, hipMemcpyKind::hipMemcpyHostToDevice, stream0);
		hipMemcpyAsync(db, &b[dataOffset], MEMSIZE, hipMemcpyKind::hipMemcpyHostToDevice, stream0);
		//TODO:  run the kernel in the stream
		kernel << <GRID_SIZE, THREAD_PER_BLOCK, 0, stream0 >> > (da, db, MEMSIZE, dc);
		//TODO:  copy dc->c
		hipMemcpyAsync(&c[dataOffset], dc, MEMSIZE, hipMemcpyKind::hipMemcpyDeviceToHost, stream0);
		dataOffset += N;

		hipMemcpyAsync(da, &a[dataOffset], MEMSIZE, hipMemcpyKind::hipMemcpyHostToDevice, stream1);
		hipMemcpyAsync(db, &b[dataOffset], MEMSIZE, hipMemcpyKind::hipMemcpyHostToDevice, stream1);
		//TODO:  run the kernel in the stream
		kernel << <GRID_SIZE, THREAD_PER_BLOCK, 0, stream1 >> > (da, db, MEMSIZE, dc);
		//TODO:  copy dc->c
		hipMemcpyAsync(&c[dataOffset], dc, MEMSIZE, hipMemcpyKind::hipMemcpyDeviceToHost, stream1);
		dataOffset += N;
	}

	////TODO: Synchonize stream
	hipStreamSynchronize(stream0);
	hipStreamSynchronize(stream1);
	////TODO: Destroy stream
	checkCudaErrors(hipStreamDestroy(stream0));
	checkCudaErrors(hipStreamDestroy(stream1));

	printData(a, 10);
	printData(b, 10);
	printData(c, 10);

	hipFree(da);
	hipFree(db);
	hipFree(dc);

	hipHostFree(a);
	hipHostFree(b);
	hipHostFree(c);
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Tests 3. - two streams - breadth first approach</summary>
///
/// <remarks>	Gajdi, 16. 4. 2013. </remarks>
////////////////////////////////////////////////////////////////////////////////////////////////////
void test3()
{
	//TODO: reuse the source code of above mentioned method test1()
	unsigned int *a, *b, *c;
	unsigned int *da, *db, *dc;

	// paged-locked allocation
	hipHostAlloc((void**)&a, NO_LOOPS * MEMSIZE, hipHostMallocDefault);
	hipHostAlloc((void**)&b, NO_LOOPS * MEMSIZE, hipHostMallocDefault);
	hipHostAlloc((void**)&c, NO_LOOPS * MEMSIZE, hipHostMallocDefault);

	fillData(a, NO_LOOPS * N);
	fillData(b, NO_LOOPS * N);

	// Data chunks on GPU
	hipMalloc((void**)&da, MEMSIZE);
	hipMalloc((void**)&db, MEMSIZE);
	hipMalloc((void**)&dc, MEMSIZE);

	////TODO: create stream
	hipStream_t stream0;
	checkCudaErrors(hipStreamCreate(&stream0));
	hipStream_t stream1;
	checkCudaErrors(hipStreamCreate(&stream1));

	unsigned int dataOffset = 0;
	for (int i = 0; i < NO_LOOPS / 2; i++)
	{
		//TODO:  copy a->da, b->db
		hipMemcpyAsync(da, &a[dataOffset], MEMSIZE, hipMemcpyKind::hipMemcpyHostToDevice, stream0);
		hipMemcpyAsync(da, &a[dataOffset], MEMSIZE, hipMemcpyKind::hipMemcpyHostToDevice, stream1);
		hipMemcpyAsync(db, &b[dataOffset], MEMSIZE, hipMemcpyKind::hipMemcpyHostToDevice, stream0);
		hipMemcpyAsync(db, &b[dataOffset], MEMSIZE, hipMemcpyKind::hipMemcpyHostToDevice, stream1);

		//TODO:  run the kernel in the stream
		kernel << <GRID_SIZE, THREAD_PER_BLOCK, 0, stream0 >> > (da, db, MEMSIZE, dc);
		kernel << <GRID_SIZE, THREAD_PER_BLOCK, 0, stream1 >> > (da, db, MEMSIZE, dc);

		//TODO:  copy dc->c
		hipMemcpyAsync(&c[dataOffset], dc, MEMSIZE, hipMemcpyKind::hipMemcpyDeviceToHost, stream0);
		dataOffset += N;
		hipMemcpyAsync(&c[dataOffset], dc, MEMSIZE, hipMemcpyKind::hipMemcpyDeviceToHost, stream1);
		dataOffset += N;
	}

	////TODO: Synchonize stream
	hipStreamSynchronize(stream0);
	hipStreamSynchronize(stream1);
	////TODO: Destroy stream
	checkCudaErrors(hipStreamDestroy(stream0));
	checkCudaErrors(hipStreamDestroy(stream1));

	printData(a, 10);
	printData(b, 10);
	printData(c, 10);

	hipFree(da);
	hipFree(db);
	hipFree(dc);

	hipHostFree(a);
	hipHostFree(b);
	hipHostFree(c);
}

int main(int argc, char *argv[])
{
	initializeCUDA(deviceProp);

	hipEvent_t startEvent, stopEvent;
	float elapsedTime;

	hipEventCreate(&startEvent);
	hipEventCreate(&stopEvent);

	hipEventRecord(startEvent, 0);
	test1();
	hipEventRecord(stopEvent, 0);
	hipEventSynchronize(stopEvent);
	hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
	printf("Test time: %f ms\n", elapsedTime);

	hipEventRecord(startEvent, 0);
	test2();
	hipEventRecord(stopEvent, 0);
	hipEventSynchronize(stopEvent);
	hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
	printf("Test time: %f ms\n", elapsedTime);

	hipEventRecord(startEvent, 0);
	test3();
	hipEventRecord(stopEvent, 0);
	hipEventSynchronize(stopEvent);
	hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
	printf("Test time: %f ms\n", elapsedTime);

	hipEventDestroy(startEvent);
	hipEventDestroy(stopEvent);

	return 0;
}
