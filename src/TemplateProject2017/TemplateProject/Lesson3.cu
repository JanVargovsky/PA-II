#include "hip/hip_runtime.h"
#include <cudaDefs.h>
#include <time.h>
#include <math.h>
#include <random>

namespace lesson3 {

	//WARNING!!! Do not change TPB and NO_FORCES for this demo !!!
	constexpr unsigned int TPB = 128;
	constexpr unsigned int NO_FORCES = 256;
	constexpr unsigned int NO_RAIN_DROPS = 1 << 20;

	constexpr unsigned int MEM_BLOCKS_PER_THREAD_BLOCK = 8;

	hipError_t error = hipSuccess;
	hipDeviceProp_t deviceProp = hipDeviceProp_t();

	using namespace std;

	random_device rd;
	float3 *createData(const unsigned int length, bool random)
	{
		//TODO: Generate float3 vectors. You can use 'make_float3' method.
		// mersenne twister
		auto mt = mt19937_64(rd());
		auto urd = uniform_real_distribution<float>(-1, 1);

		float3 *data = new float3[length];
		if (random)
			for (size_t i = 0; i < length; i++)
				data[i] = make_float3(urd(mt), urd(mt), urd(mt));
		else
			for (size_t i = 0; i < length; i++)
				data[i] = make_float3(1.f, 1.f, 1.f);
		return data;
	}

	void printData(const float3 *data, const unsigned int length)
	{
		if (data == 0) return;
		const float3 *ptr = data;
		for (unsigned int i = 0; i < length; i++, ptr++)
		{
			printf("%5.2f %5.2f %5.2f ", ptr->x, ptr->y, ptr->z);
		}
	}

	////////////////////////////////////////////////////////////////////////////////////////////////////
	/// <summary>	Sums the forces to get the final one using parallel reduction. 
	/// 		    WARNING!!! The method was written to meet input requirements of our example, i.e. 128 threads and 256 forces  </summary>
	/// <param name="dForces">	  	The forces. </param>
	/// <param name="noForces">   	The number of forces. </param>
	/// <param name="dFinalForce">	[in,out] If non-null, the final force. </param>
	////////////////////////////////////////////////////////////////////////////////////////////////////
	__global__ void reduce(const float3 * __restrict__ dForces, const unsigned int noForces, float3* __restrict__ dFinalForce)
	{
		__shared__ float3 sForces[TPB];					//SEE THE WARNING MESSAGE !!!
		unsigned int tid = threadIdx.x;
		unsigned int next = TPB;						//SEE THE WARNING MESSAGE !!!

		//TODO: Make the reduction
		if (tid >= noForces)
			return;

		float3 *src = &sForces[tid];
		float3 *src2 = (float3*)&dForces[tid + next];
		// global memory -> shared memory
		*src = dForces[tid];
		src->x += src2->x;
		src->y += src2->y;
		src->z += src2->z;
		__syncthreads();

		next >>= 1; // 64
		if (tid >= next) return;
		src2 = src + next;
		src->x += src2->x;
		src->y += src2->y;
		src->z += src2->z;
		__syncthreads();

		next >>= 1; // 32
		if (tid >= next) return;
		src2 = src + next;
		src->x += src2->x;
		src->y += src2->y;
		src->z += src2->z;

		next >>= 1; // 16
		if (tid >= next) return;
		src2 = src + next;
		src->x += src2->x;
		src->y += src2->y;
		src->z += src2->z;

		next >>= 1; // 8
		if (tid >= next) return;
		src2 = src + next;
		src->x += src2->x;
		src->y += src2->y;
		src->z += src2->z;

		next >>= 1; // 4
		if (tid >= next) return;
		src2 = src + next;
		src->x += src2->x;
		src->y += src2->y;
		src->z += src2->z;

		next >>= 1; // 2
		if (tid >= next) return;
		src2 = src + next;
		src->x += src2->x;
		src->y += src2->y;
		src->z += src2->z;

		next >>= 1; // 1
		if (tid >= next) return;
		src2 = src + next;
		src->x += src2->x;
		src->y += src2->y;
		src->z += src2->z;

		if (tid == 0)
			// shared memory -> global memory
			*dFinalForce = src[0];
	}

	////////////////////////////////////////////////////////////////////////////////////////////////////
	/// <summary>	Adds the FinalForce to every Rain drops position. </summary>
	/// <param name="dFinalForce">	The final force. </param>
	/// <param name="noRainDrops">	The number of rain drops. </param>
	/// <param name="dRainDrops"> 	[in,out] If non-null, the rain drops positions. </param>
	////////////////////////////////////////////////////////////////////////////////////////////////////
	__global__ void add(const float3* __restrict__ dFinalForce, const unsigned int noRainDrops, float3* __restrict__ dRainDrops)
	{
		//TODO: Add the FinalForce to every Rain drops position.
		unsigned int bid = blockIdx.x * MEM_BLOCKS_PER_THREAD_BLOCK + threadIdx.x;
#pragma unroll MEM_BLOCKS_PER_THREAD_BLOCK
		for (size_t i = 0; i < MEM_BLOCKS_PER_THREAD_BLOCK; i++)
		{
			auto tid = bid + i;
			if (tid >= noRainDrops)
				return;
			dRainDrops[tid].x += dFinalForce->x;
			dRainDrops[tid].y += dFinalForce->y;
			dRainDrops[tid].z += dFinalForce->z;
		}
	}

	void run()
	{
		initializeCUDA(deviceProp);

		hipEvent_t startEvent, stopEvent;
		float elapsedTime;

		hipEventCreate(&startEvent);
		hipEventCreate(&stopEvent);
		hipEventRecord(startEvent, 0);

		float3 *hForces = createData(NO_FORCES, true);
		float3 *hDrops = createData(NO_RAIN_DROPS, false);

		float3 *dForces = nullptr;
		float3 *dDrops = nullptr;
		float3 *dFinalForce = nullptr;

		error = hipMalloc((void**)&dForces, NO_FORCES * sizeof(float3));
		error = hipMemcpy(dForces, hForces, NO_FORCES * sizeof(float3), hipMemcpyHostToDevice);

		error = hipMalloc((void**)&dDrops, NO_RAIN_DROPS * sizeof(float3));
		error = hipMemcpy(dDrops, hDrops, NO_RAIN_DROPS * sizeof(float3), hipMemcpyHostToDevice);

		error = hipMalloc((void**)&dFinalForce, sizeof(float3));

		KernelSetting ksReduce;
		//TODO: ... Set ksReduce
		ksReduce.dimBlock = dim3(TPB, 1, 1);
		ksReduce.dimGrid = dim3(1, 1, 1);


		KernelSetting ksAdd;
		//TODO: ... Set ksAdd
		ksAdd.dimBlock = dim3(TPB, 1, 1);
		ksAdd.dimGrid = dim3(getNumberOfParts(NO_RAIN_DROPS, TPB * MEM_BLOCKS_PER_THREAD_BLOCK), 1, 1);

		reduce << <ksReduce.dimGrid, ksReduce.dimBlock >> > (dForces, NO_FORCES, dFinalForce);
		checkDeviceMatrix<float>((float*)dFinalForce, sizeof(float3), 1, 3, "%5.2f ", "Final force");

		for (unsigned int i = 0; i < 1000; i++)
		{
			reduce << <ksReduce.dimGrid, ksReduce.dimBlock >> > (dForces, NO_FORCES, dFinalForce);
			add << <ksAdd.dimGrid, ksAdd.dimBlock >> > (dFinalForce, NO_RAIN_DROPS, dDrops);
			checkDeviceMatrix<float>((float*)dDrops, sizeof(float3), 1, 3, "%5.2f ", "Final Rain Drops");
		}

		checkDeviceMatrix<float>((float*)dFinalForce, sizeof(float3), 1, 3, "%5.2f ", "Final force");
		checkDeviceMatrix<float>((float*)dDrops, sizeof(float3), NO_RAIN_DROPS, 3, "%5.2f ", "Final Rain Drops");


		if (hForces)
			free(hForces);
		if (hDrops)
			free(hDrops);

		hipFree(dForces);
		hipFree(dDrops);

		hipEventRecord(stopEvent, 0);
		hipEventSynchronize(stopEvent);

		hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
		hipEventDestroy(startEvent);
		hipEventDestroy(stopEvent);

		printf("Time to get device properties: %f ms", elapsedTime);
	}

}