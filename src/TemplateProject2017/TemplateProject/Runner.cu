#include <cudaDefs.h>
#include "Lesson1.h";

hipDeviceProp_t deviceProp = hipDeviceProp_t();

int main(int argc, char *argv[])
{
	initializeCUDA(deviceProp);
	lesson1::run();
	return 0;
}
